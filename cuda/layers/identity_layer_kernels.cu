#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

extern "C" {
#include "layers/identity_layer.h"
#include "utils/blas.h"
#include "dark_cuda.h"
}


extern "C" void forward_identity_layer_gpu(const layer l, network_state net) {
    hipMemcpy(l.output_gpu, net.input, l.batch * l.inputs, hipMemcpyDeviceToDevice);
    check_error(hipPeekAtLastError());
}

extern "C" void backward_identity_layer_gpu(layer l, network_state net) {}
