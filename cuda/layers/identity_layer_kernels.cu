#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hipblas.h>

extern "C" {
#include "layers/identity_layer.h"
#include "dark_cuda.h"
}


extern "C" void forward_identity_layer_gpu(const layer l, network_state net) {
    check_error(hipMemcpy(l.output_gpu, net.input, l.batch * l.inputs * sizeof(float), hipMemcpyDeviceToDevice));
}

extern "C" void backward_identity_layer_gpu(layer l, network_state net) {}
